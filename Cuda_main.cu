#include "hip/hip_runtime.h"
#include "Header.h"
#include "math.h"

__device__ double HLLD_Alexashov(const double& ro_L, const double& p_L, const double& v1_L, const double& v2_L, const double& v3_L,//
    const double& Bx_L, const double& By_L, const double& Bz_L, const double& ro_R, const double& p_R, const double& v1_R, const double& v2_R, const double& v3_R,//
    const double& Bx_R, const double& By_R, const double& Bz_R, double* P, const double& n1, const double& n2, const double& n3, double& rad, int metod = 0);

__device__ double minmod(double x, double y)
{
    if (sign(x) + sign(y) == 0)
    {
        return 0.0;
    }
    else
    {
        return   ((sign(x) + sign(y)) / 2.0) * min(fabs(x), fabs(y));  ///minmod
        //return (2*x*y)/(x + y);   /// vanleer
    }
}

__device__ double linear(double x1, double t1, double x2, double t2, double x3, double t3, double y)
{
    double d = minmod((t1 - t2) / (x1 - x2), (t2 - t3) / (x2 - x3));
    return  (d * (y - x2) + t2);
}

__device__ void linear2(double x1, double t1, double x2, double t2, double x3, double t3, double y1, double y2,//
    double& A, double& B)
{
    // ������� �������� - ����������� - �� ������� �� ����
    double d = minmod((t1 - t2) / (x1 - x2), (t2 - t3) / (x2 - x3));
    A = (d * (y1 - x2) + t2);
    B = (d * (y2 - x2) + t2);
    //printf("%lf | %lf | %lf | %lf | %lf | %lf | %lf | %lf | %lf | %lf \n", x1, t1, x2, t2, x3, t3, y1, y2, A, B);
    return;
}

__device__ int sign(double& x)
{
    if (x > 0)
    {
        return 1;
    }
    else if (x < 0)
    {
        return  -1;
    }
    else
    {
        return 0;
    }
}

__device__ void f_TVD(double& dx, double& p1, double& p2, double& p3, double& p4, double& p12, double& p21, double& s1, double& s2, double& s3)
{
    //double s1 = __dsqrt_rn(kv(x1 - x3) + kv(y1 - y3) + kv(z1 - z3));
    //double s2 = __dsqrt_rn(kv(x1 - x2) + kv(y1 - y2) + kv(z1 - z2));
    //double s3 = __dsqrt_rn(kv(x4 - x2) + kv(y4 - y2) + kv(z4 - z2));

    p12 = linear(-s1, p3, 0.0, p1, s2, p2, dx);
    p21 = linear(0.0, p1, s2, p2, s2 + s3, p4, dx);
}


//  void chlld(id_bn, n_state, n_disco, KOBL, i_in, j_in, k_in, kdir, al, be, ge, el, w, qqq1, qqq2, dsl, dsp, dsc, ythll, qqq)
__device__ double HLLD_Alexashov(const double& ro_L, const double& p_L, const double& v1_L, const double& v2_L, const double& v3_L,//
    const double& Bx_L, const double& By_L, const double& Bz_L, const double& ro_R, const double& p_R, const double& v1_R, const double& v2_R, const double& v3_R,//
    const double& Bx_R, const double& By_R, const double& Bz_R, double* P, const double& n1, const double& n2, const double& n3, double& rad, int metod)
{
    int x0 = 0, x1 = 1, x2 = 2;
    double aco[3][3];
    int n_state = metod;
    //c-------  n_state=0   - one speed LAX
    //c-------  n_state=1   - two speed LAX (HLL,(Harten-Lax-van-Leer))
    //c-------  n_state=2   - two-state (3 speed) HLLC (Contact Discontinuity)
    //c-------  n_state=3   - multi-state (5 speed) HLLD (All Discontinuity)

    double FR[8], FL[8], dq[8];
    double FW[8], UL[8], UZ[8], UR[8];
     double UZL[8], UZR[8];
     double UZZL[8], UZZR[8];
    double vL[3], vR[3], bL[3], bR[3];
     double vzL[3], vzR[3], bzL[3], bzR[3];
     double vzzL[3], vzzR[3], bzzL[3], bzzR[3];
    double qv[3], qb[3];



    double wv = 0.0;
    int n_disco = 0; // ��� ����������� ��������� �������������


    double r1 = ro_L;
    double u1 = v1_L;
    double v1 = v2_L;
    double w1 = v3_L;
    double p1 = p_L;
    double bx1 = Bx_L / spi4;
    double by1 = By_L / spi4;
    double bz1 = Bz_L / spi4;


    double r2 = ro_R;
    double u2 = v1_R;
    double v2 = v2_R;
    double w2 = v3_R;
    double p2 = p_R;
    double bx2 = Bx_R / spi4;
    double by2 = By_R / spi4;
    double bz2 = Bz_R / spi4;

    double ro = (r2 + r1) / x2;
    double au = (u2 + u1) / x2;
    double av = (v2 + v1) / x2;
    double aw = (w2 + w1) / x2;
    double ap = (p2 + p1) / x2;
    double abx = (bx2 + bx1) / x2;
    double aby = (by2 + by1) / x2;
    double abz = (bz2 + bz1) / x2;

    double al = n1;
    double be = n2;
    double ge = n3;

    double bk = abx * al + aby * be + abz * ge;
    double b2 = kv(abx) + kv(aby) + kv(abz);

    double  d = b2 - kv(bk);
    aco[0][0] = al;
    aco[1][0] = be;
    aco[2][0] = ge;

    if (d > 0.00001)
    {
        d = __dsqrt_rn(d);
        aco[0][1] = (abx - bk * al) / d;
        aco[1][1] = (aby - bk * be) / d;
        aco[2][1] = (abz - bk * ge) / d;
        aco[0][2] = (aby * ge - abz * be) / d;
        aco[1][2] = (abz * al - abx * ge) / d;
        aco[2][2] = (abx * be - aby * al) / d;
    }
    else
    {
        double aix, aiy, aiz;
        if ( (fabs(al) < fabs(be)) && (fabs(al) < fabs(ge)) )
        {
            aix = x1;
            aiy = x0;
            aiz = x0;
        }
        else if ( fabs(be) < fabs(ge) )
        {
            aix = x0;
            aiy = x1;
            aiz = x0;
        }
        else
        {
            aix = x0;
            aiy = x0;
            aiz = x1;
        }
        double aik = aix * al + aiy * be + aiz * ge;
        d = __dsqrt_rn(x1 - kv(aik));
        aco[0][1] = (aix - aik * al) / d;
        aco[1][1] = (aiy - aik * be) / d;
        aco[2][1] = (aiz - aik * ge) / d;
        aco[0][2] = (aiy * ge - aiz * be) / d;
        aco[1][2] = (aiz * al - aix * ge) / d;
        aco[2][2] = (aix * be - aiy * al) / d;
    }

    aco[0][0] = al;
    aco[1][0] = be;
    aco[2][0] = ge;

    //if (fabs(skk(aco[0][0], aco[1][0], aco[2][0], aco[0][1], aco[1][1], aco[2][1])) > 0.000001 || //
    //    fabs(skk(aco[0][0], aco[1][0], aco[2][0], aco[0][2], aco[1][2], aco[2][2])) > 0.000001 || //
    //    fabs(skk(aco[0][2], aco[1][2], aco[2][2], aco[0][1], aco[1][1], aco[2][1])) > 0.000001 || //
    //    fabs(kvv(aco[0][0], aco[1][0], aco[2][0]) - 1.0) > 0.000001 || fabs(kvv(aco[0][1], aco[1][1], aco[2][1]) - 1.0) > 0.000001 ||//
    //    fabs(kvv(aco[0][2], aco[1][2], aco[2][2]) - 1.0) > 0.000001)
    //{
    //    printf("Ne normal  174fdcdsaxes\n");
    //}


    for (int i = 0; i < 3; i++)
    {
        vL[i] = aco[0][i] * u1 + aco[1][i] * v1 + aco[2][i] * w1;
        vR[i] = aco[0][i] * u2 + aco[1][i] * v2 + aco[2][i] * w2;
        bL[i] = aco[0][i] * bx1 + aco[1][i] * by1 + aco[2][i] * bz1;
        bR[i] = aco[0][i] * bx2 + aco[1][i] * by2 + aco[2][i] * bz2;
    }



    double aaL = bL[0] / __dsqrt_rn(r1);
    double b2L = kv(bL[0]) + kv(bL[1]) + kv(bL[2]);
    double b21 = b2L / r1;
    double cL = __dsqrt_rn(ga * p1 / r1);
    double qp = __dsqrt_rn(b21 + cL * (cL + 2.0 * aaL));
    double qm = __dsqrt_rn(b21 + cL * (cL - 2.0 * aaL));
    double cfL = (qp + qm) / x2;
    double ptL = p1 + b2L / x2;

    double aaR = bR[0] / __dsqrt_rn(r2);
    double b2R = kv(bR[0]) + kv(bR[1]) + kv(bR[2]);
    double b22 = b2R / r2;
    double cR = __dsqrt_rn(ga * p2 / r2);
    qp = __dsqrt_rn(b22 + cR * (cR + 2.0 * aaR));
    qm = __dsqrt_rn(b22 + cR * (cR - 2.0 * aaR));
    double cfR = (qp + qm) / x2;
    double ptR = p2 + b2R / x2;

    double aC = (aaL + aaR) / x2;
    double b2o = (b22 + b21) / x2;
    double cC = __dsqrt_rn(ga * ap / ro);
    qp = __dsqrt_rn(b2o + cC * (cC + x2 * aC));
    qm = __dsqrt_rn(b2o + cC * (cC - x2 * aC));
    double cfC = (qp + qm) / x2;
    double vC1 = (vL[0] + vR[0]) / x2;

    double SL, SR;

    if(true)
    {
        SL = min(vL[0], vR[0]) - max(cfL, cfR);
        SR = max(vL[0], vR[0]) + max(cfL, cfR);
    }
    else if (n_disco == 1)
    {
        SL = min((vL[0] - cfL), (vC1 - cfC));
        SR = max((vR[0] + cfR), (vC1 + cfC));
    }
    else if (n_disco == 0)
    {
        SL = min((vL[0] - cfL), (vR[0] - cfR));
        SR = max((vL[0] + cfL), (vR[0] + cfR));
    }
    else if (n_disco == 2)
    {
        double SL_1 = min((vL[0] - cfL), (vC1 - cfC));
        double SR_1 = max((vR[0] + cfR), (vC1 + cfC));
        double SL_2 = min((vL[0] - cfL), (vR[0] - cfR));
        double SR_2 = max((vL[0] + cfL), (vR[0] + cfR));
        double oo = 0.75;
        double oo1 = 1.0 - oo;
        SL = oo * SL_1 + oo1 * SL_2;
        SR = oo * SR_1 + oo1 * SR_2;
    }


    double suR = SR - vR[0];
    double suL = SL - vL[0];
    double SM = (suR * r2 * vR[0] - ptR + ptL - suL * r1 * vL[0]) / (suR * r2 - suL * r1);

    // double dsl = SL;
    // double dsc = SM;
    // double dsp = SR;

    if ( (SR < SL)||(SL > SM)||(SR < SM) )
    {
        printf("ERROR -  254 fghrvtrgr\n");
        printf("%lf, %lf, %lf, %lf, %lf, %lf, %lf, %lf, %lf, %lf\n",//
            vL[0], vR[0], cfL, cfR, ro_L, ro_R, p_L, p_R, suR, suL);
    }

    double UU = max(fabs(SL), fabs(SR));
    double time = krit * rad / UU;

    double TR0, TL0;
    if (n_state == 0)
    {
        TR0 = fabs(vL[0] + vR[0]) / x2 + cfC;
        TL0 = -TR0;
        SR = TR0;
        SL = TL0;
    }


    double upt1 = (kv(u1) + kv(v1) + kv(w1)) / 2.0;
    double sbv1 = u1 * bx1 + v1 * by1 + w1 * bz1;

    double upt2 = (kv(u2) + kv(v2) + kv(w2)) / 2.0;
    double sbv2 = u2 * bx2 + v2 * by2 + w2 * bz2;

    double e1 = p1 / g1 + r1 * upt1 + b2L / x2;
    double e2 = p2 / g1 + r2 * upt2 + b2R / x2;

    FL[0] = r1 * vL[0];
    FL[1] = r1 * vL[0] * vL[0] + ptL - kv(bL[0]);
    FL[2] = r1 * vL[0] * vL[1] - bL[0] * bL[1];
    FL[3] = r1 * vL[0] * vL[2] - bL[0] * bL[2];
    FL[4] = (e1 + ptL) * vL[0] - bL[0] * sbv1;
    FL[5] = 0.0;
    FL[6] = vL[0] * bL[1] - vL[1] * bL[0];
    FL[7] = vL[0] * bL[2] - vL[2] * bL[0];

    FR[0] = r2 * vR[0];
    FR[1] = r2 * vR[0] * vR[0] + ptR - kv(bR[0]);
    FR[2] = r2 * vR[0] * vR[1] - bR[0] * bR[1];
    FR[3] = r2 * vR[0] * vR[2] - bR[0] * bR[2];
    FR[4] = (e2 + ptR) * vR[0] - bR[0] * sbv2;
    FR[5] = 0.0;
    FR[6] = vR[0] * bR[1] - vR[1] * bR[0];
    FR[7] = vR[0] * bR[2] - vR[2] * bR[0];

    UL[0] = r1;
    UL[4] = e1;
    UR[0] = r2;
    UR[4] = e2;

    for (int i = 0; i < 3; i++)
    {

        UL[i + 1] = r1 * vL[i];
        UL[i + 5] = bL[i];
        UR[i + 1] = r2 * vR[i];
        UR[i + 5] = bR[i];
    }

    for (int ik = 0; ik < 8; ik++)
    {
        UZ[ik] = (SR * UR[ik] - SL * UL[ik] + FL[ik] - FR[ik]) / (SR - SL);
    }


    if (n_state <= 1)
    {

        for (int ik = 0; ik < 8; ik++)
        {
            dq[ik] = UR[ik] - UL[ik];
        }



        double TL = SL;
        double TR = SR;
        if (SL > wv)
        {
            TL = 0.0;
            for (int ik = 0; ik < 8; ik++)
            {
                FW[ik] = wv * UL[ik];
            }
        }
        else if ( (SL <= wv) && (wv <= SR) )
        {
            for (int ik = 0; ik < 8; ik++)
            {
                FW[ik] = wv * UZ[ik];
            }
        }
        else if (SR < wv)
        {
            TR = 0.0;
            for (int ik = 0; ik < 8; ik++)
            {
                FW[ik] = wv * UR[ik];
            }
        }
        else
        {
            printf("ERROR  329 87732, %lf, %lf, %lf, %lf, %lf, %lf, %lf\n", r1, r2, p1, p2, al, be, ge);
        }


        double a = TR * TL;
        double b = TR - TL;

        P[0] = (TR * FL[0] - TL * FR[0] + a * dq[0]) / b - FW[0];
        P[4] = (TR * FL[4] - TL * FR[4] + a * dq[4]) / b - FW[4];

        for (int ik = 1; ik < 4; ik++)
        {
            qv[ik - 1] = (TR * FL[ik] - TL * FR[ik] + a * dq[ik]) / b - FW[ik];
        }

        for (int ik = 5; ik < 8; ik++)
        {
            qb[ik - 5] = (TR * FL[ik] - TL * FR[ik] + a * dq[ik]) / b - FW[ik];
        }

        double SN = max(fabs(SL), fabs(SR));

        double wbn = 0.0;
        if (wv >= SR)
        {
            wbn = wv * bR[0];
        }
        else if (wv <= SL)
        {
            wbn = wv * bL[0];
        }
        else
        {
            wbn = wv * (bL[0] + bR[0]) / x2;
        }

        qb[0] = -SN * (bR[0] - bL[0]) - wbn;

        for (int ik = 0; ik < 3; ik++)
        {
            P[ik + 1] = aco[ik][0] * qv[0] + aco[ik][1] * qv[1] + aco[ik][2] * qv[2];
            P[ik + 5] = aco[ik][0] * qb[0] + aco[ik][1] * qb[1] + aco[ik][2] * qb[2];
            P[ik + 5] = spi4 * P[ik + 5];
        }

        double SWAP = P[4];
        P[4] = P[5];
        P[5] = P[6];
        P[6] = P[7];
        P[7] = SWAP;
        return time;
    }
    if (n_state == 3)
    {
         
        double ptz = (suR * r2 * ptL - suL * r1 * ptR + r1 * r2 * suR * suL * (vR[0] - vL[0])) / (suR * r2 - suL * r1);

        vzL[0] = SM;
        vzR[0] = SM;
        vzzL[0] = SM;
        vzzR[0] = SM;
        double ptzL = ptz;
        double ptzR = ptz;
        double ptzzL = ptz;
        double ptzzR = ptz;

        double suRm = suR / (SR - SM);
        double suLm = suL / (SL - SM);
        double rzR = r2 * suRm;
        double rzL = r1 * suLm;

        double bn = UZ[5];
        double bn2 = bn * bn;
        bzL[0] = bn;
        bzR[0] = bn;
        bzzL[0] = bn;
        bzzR[0] = bn;

        double ttR = r2 * suR * (SR - SM) - bn2;
        double tvR, tbR, tvL, tbL;
        if (fabs(ttR) <= 0.00000001)
        {
            tvR = x0;
            tbR = x0;
        }
        else
        {
            tvR = (SM - vR[0]) / ttR;
            tbR = (r2 * suR * suR - bn2) / ttR;
        }

        double ttL = r1 * suL * (SL - SM) - bn2;
        if (fabs(ttL) <= 0.00000001)
        {
            tvL = x0;
            tbL = x0;
        }
        else
        {
            tvL = (SM - vL[0]) / ttL;
            tbL = (r1 * suL * suL - bn2) / ttL;
        }

        vzL[1] = vL[1] - bn * bL[1] * tvL;
        vzL[2] = vL[2] - bn * bL[2] * tvL;
        vzR[1] = vR[1] - bn * bR[1] * tvR;
        vzR[2] = vR[2] - bn * bR[2] * tvR;

        bzL[1] = bL[1] * tbL;
        bzL[2] = bL[2] * tbL;
        bzR[1] = bR[1] * tbR;
        bzR[2] = bR[2] * tbR;

        double sbvL = bzL[0] * vzL[0] + bzL[1] * vzL[1] + bzL[2] * vzL[2];
        double sbvR = bzR[0] * vzR[0] + bzR[1] * vzR[1] + bzR[2] * vzR[2];

        double ezR = e2 * suRm + (ptz * SM - ptR * vR[0] + bn * (sbv2 - sbvR)) / (SR - SM);
        double ezL = e1 * suLm + (ptz * SM - ptL * vL[0] + bn * (sbv1 - sbvL)) / (SL - SM);

        double rzzR = rzR;
        double rzzL = rzL;
        double rzRs = __dsqrt_rn(rzR);
        double rzLs = __dsqrt_rn(rzL);
        double rzss = rzRs + rzLs;
        double rzps = rzRs * rzLs;

        double SZL = SM - fabs(bn) / rzLs;
        double SZR = SM + fabs(bn) / rzRs;

        int ibn = 0;
        double sbn;
        if (fabs(bn) > 0.000001)
        {
            sbn = 1.0 * sign(bn);
            ibn = 1;
        }
        else
        {
            sbn = 0.0;
            ibn = 0;
            SZL = SM;
            SZR = SM;
        }

        vzzL[1] = (rzLs * vzL[1] + rzRs * vzR[1] + sbn * (bzR[1] - bzL[1])) / rzss;
        vzzL[2] = (rzLs * vzL[2] + rzRs * vzR[2] + sbn * (bzR[2] - bzL[2])) / rzss;
        vzzR[1] = vzzL[1];
        vzzR[2] = vzzL[2];

        bzzL[1] = (rzLs * bzR[1] + rzRs * bzL[1] + sbn * rzps * (vzR[1] - vzL[1])) / rzss;
        bzzL[2] = (rzLs * bzR[2] + rzRs * bzL[2] + sbn * rzps * (vzR[2] - vzL[2])) / rzss;
        bzzR[1] = bzzL[1];
        bzzR[2] = bzzL[2];

        double sbzz = bzzL[0] * vzzL[0] + bzzL[1] * vzzL[1] + bzzL[2] * vzzL[2];

        double ezzR = ezR + rzRs * sbn * (sbvR - sbzz);
        double ezzL = ezL - rzLs * sbn * (sbvL - sbzz);

        UZL[0] = rzL;
        UZL[4] = ezL;
        UZR[0] = rzR;
        UZR[4] = ezR;

        for (int ik = 0; ik < 3; ik++)
        {
            UZL[ik + 1] = vzL[ik] * rzL;
            UZL[ik + 5] = bzL[ik];
            UZR[ik + 1] = vzR[ik] * rzR;
            UZR[ik + 5] = bzR[ik];
        }



        UZZL[0] = rzzL;
        UZZL[4] = ezzL;
        UZZR[0] = rzzR;
        UZZR[4] = ezzR;
        for (int ik = 0; ik < 3; ik++)
        {
            UZZL[ik + 1] = vzzL[ik] * rzzL;
            UZZL[ik + 5] = bzzL[ik];
            UZZR[ik + 1] = vzzR[ik] * rzzR;
            UZZR[ik + 5] = bzzR[ik];
        }



        if (SL > wv)
        {
            P[0] = FL[0] - wv * UL[0];
            P[4] = FL[4] - wv * UL[4];
            for (int ik = 1; ik < 4; ik++)
            {
                qv[ik - 1] = FL[ik] - wv * UL[ik];
            }

            for (int ik = 5; ik < 8; ik++)
            {
                qb[ik - 5] = FL[ik] - wv * UL[ik];
            }
        }

        if ( (SL <= wv) && (SZL >= wv) )
        {
            int ik = 0;
            P[ik] = FL[ik] + SL * (UZL[ik] - UL[ik]) - wv * UZL[ik];
            ik = 4;
            P[ik] = FL[ik] + SL * (UZL[ik] - UL[ik]) - wv * UZL[ik];
            for (int ik = 1; ik < 4; ik++)
            {
                qv[ik - 1] = FL[ik] + SL * (UZL[ik] - UL[ik]) - wv * UZL[ik];
            }

            for (int ik = 5; ik < 8; ik++)
            {
                qb[ik - 5] = FL[ik] + SL * (UZL[ik] - UL[ik]) - wv * UZL[ik];
            }
        }
        //c------ FZZ
        if (ibn == 1)
        {

            if ( (SZL <= wv) && (SM >= wv) )
            {
                int ik = 0;
                P[ik] = FL[ik] + SZL * (UZZL[ik] - UZL[ik]) + SL * (UZL[ik] - UL[ik]) - wv * UZZL[ik];
                ik = 4;
                P[ik] = FL[ik] + SZL * (UZZL[ik] - UZL[ik]) + SL * (UZL[ik] - UL[ik]) - wv * UZZL[ik];
                for (int ik = 1; ik < 4; ik++)
                {
                    qv[ik - 1] = FL[ik] + SZL * (UZZL[ik] - UZL[ik]) + SL * (UZL[ik] - UL[ik]) - wv * UZZL[ik];
                }

                for (int ik = 5; ik < 8; ik++)
                {
                    qb[ik - 5] = FL[ik] + SZL * (UZZL[ik] - UZL[ik]) + SL * (UZL[ik] - UL[ik]) - wv * UZZL[ik];
                }
            }

            if ( (SM <= wv) && (SZR >= wv) )
            {
                int ik = 0;
                P[ik] = FR[ik] + SZR * (UZZR[ik] - UZR[ik]) + SR * (UZR[ik] - UR[ik]) - wv * UZZR[ik];
                ik = 4;
                P[ik] = FR[ik] + SZR * (UZZR[ik] - UZR[ik]) + SR * (UZR[ik] - UR[ik]) - wv * UZZR[ik];
                for (int ik = 1; ik < 4; ik++)
                {
                    qv[ik - 1] = FR[ik] + SZR * (UZZR[ik] - UZR[ik]) + SR * (UZR[ik] - UR[ik]) - wv * UZZR[ik];
                }

                for (int ik = 5; ik < 8; ik++)
                {
                    qb[ik - 5] = FR[ik] + SZR * (UZZR[ik] - UZR[ik]) + SR * (UZR[ik] - UR[ik]) - wv * UZZR[ik];
                }
            }

        }
        //c------ 
        if ( (SZR <= wv) && (SR >= wv) )
        {
            int ik = 0;
            P[ik] = FR[ik] + SR * (UZR[ik] - UR[ik]) - wv * UZR[ik];
            ik = 4;
            P[ik] = FR[ik] + SR * (UZR[ik] - UR[ik]) - wv * UZR[ik];

            for (int ik = 1; ik < 4; ik++)
            {
                qv[ik - 1] = FR[ik] + SR * (UZR[ik] - UR[ik]) - wv * UZR[ik];
            }

            for (int ik = 5; ik < 8; ik++)
            {
                qb[ik - 5] = FR[ik] + SR * (UZR[ik] - UR[ik]) - wv * UZR[ik];
            }
        }

        if (SR < wv)
        {
            P[0] = FR[0] - wv * UR[0];
            P[4] = FR[4] - wv * UR[4];
            for (int ik = 1; ik < 4; ik++)
            {
                qv[ik - 1] = FR[ik] - wv * UR[ik];
            }


            for (int ik = 5; ik < 8; ik++)
            {
                qb[ik - 5] = FR[ik] - wv * UR[ik];
            }
        }


        //c----- Bn
        //double SN = max(fabs(SL), fabs(SR));

        double SN = max(fabs(SL), fabs(SR));

        double wbn = 0.0;
        if (wv >= SR)
        {
            wbn = wv * bR[0];
        }
        else if (wv <= SL)
        {
            wbn = wv * bL[0];
        }
        else
        {
            wbn = wv * (bL[0] + bR[0]) / x2;
        }

        qb[0] = -SN * (bR[0] - bL[0]) - wbn;

        //c-----


        for (int ik = 0; ik < 3; ik++)
        {
            P[ik + 1] = aco[ik][0] * qv[0] + aco[ik][1] * qv[1] + aco[ik][2] * qv[2];
            P[ik + 5] = aco[ik][0] * qb[0] + aco[ik][1] * qb[1] + aco[ik][2] * qb[2];
            P[ik + 5] = spi4 * P[ik + 5];
        }

        double SWAP = P[4];
        P[4] = P[5];
        P[5] = P[6];
        P[6] = P[7];
        P[7] = SWAP;
        return time;
    }
}


__device__ double HLLDQ_Alexashov(const double& ro_L, const double& Q_L, const double& p_L, const double& v1_L, const double& v2_L, const double& v3_L,//
    const double& Bx_L, const double& By_L, const double& Bz_L, const double& ro_R, const double& Q_R, const double& p_R, const double& v1_R, const double& v2_R, const double& v3_R,//
    const double& Bx_R, const double& By_R, const double& Bz_R, double* P, double& PQ, const double& n1, const double& n2, const double& n3, double& rad, int metod)
{   // �� ��������, ���� �������� ����� �� �������
    int x0 = 0, x1 = 1, x2 = 2;
    double aco[3][3];
    int n_state = metod;
    //c-------  n_state=0   - one speed LAX
    //c-------  n_state=1   - two speed LAX (HLL,(Harten-Lax-van-Leer))
    //c-------  n_state=2   - two-state (3 speed) HLLC (Contact Discontinuity)
    //c-------  n_state=3   - multi-state (5 speed) HLLD (All Discontinuity)

    double FR[8], FL[8], dq[8];
    double FW[8], UL[8], UZ[8], UR[8];
    double UZL[8], UZR[8];
    double UZZL[8], UZZR[8];
    double vL[3], vR[3], bL[3], bR[3];
    double vzL[3], vzR[3], bzL[3], bzR[3];
    double vzzL[3], vzzR[3], bzzL[3], bzzR[3];
    double qv[3], qb[3];



    double wv = 0.0;
    int n_disco = 0; // ��� ����������� ��������� �������������

    double r1 = ro_L;
    double u1 = v1_L;
    double v1 = v2_L;
    double w1 = v3_L;
    double p1 = p_L;
    double bx1 = Bx_L / spi4;
    double by1 = By_L / spi4;
    double bz1 = Bz_L / spi4;


    double r2 = ro_R;
    double u2 = v1_R;
    double v2 = v2_R;
    double w2 = v3_R;
    double p2 = p_R;
    double bx2 = Bx_R / spi4;
    double by2 = By_R / spi4;
    double bz2 = Bz_R / spi4;

    double ro = (r2 + r1) / x2;
    double au = (u2 + u1) / x2;
    double av = (v2 + v1) / x2;
    double aw = (w2 + w1) / x2;
    double ap = (p2 + p1) / x2;
    double abx = (bx2 + bx1) / x2;
    double aby = (by2 + by1) / x2;
    double abz = (bz2 + bz1) / x2;

    double al = n1;
    double be = n2;
    double ge = n3;

    double bk = abx * al + aby * be + abz * ge;
    double b2 = kv(abx) + kv(aby) + kv(abz);

    double  d = b2 - kv(bk);
    aco[0][0] = al;
    aco[1][0] = be;
    aco[2][0] = ge;

    if (d > 0.00001)
    {
        d = __dsqrt_rn(d);
        aco[0][1] = (abx - bk * al) / d;
        aco[1][1] = (aby - bk * be) / d;
        aco[2][1] = (abz - bk * ge) / d;
        aco[0][2] = (aby * ge - abz * be) / d;
        aco[1][2] = (abz * al - abx * ge) / d;
        aco[2][2] = (abx * be - aby * al) / d;
    }
    else
    {
        double aix, aiy, aiz;
        if ((fabs(al) < fabs(be)) && (fabs(al) < fabs(ge)))
        {
            aix = x1;
            aiy = x0;
            aiz = x0;
        }
        else if (fabs(be) < fabs(ge))
        {
            aix = x0;
            aiy = x1;
            aiz = x0;
        }
        else
        {
            aix = x0;
            aiy = x0;
            aiz = x1;
        }
        double aik = aix * al + aiy * be + aiz * ge;
        d = __dsqrt_rn(x1 - kv(aik));
        aco[0][1] = (aix - aik * al) / d;
        aco[1][1] = (aiy - aik * be) / d;
        aco[2][1] = (aiz - aik * ge) / d;
        aco[0][2] = (aiy * ge - aiz * be) / d;
        aco[1][2] = (aiz * al - aix * ge) / d;
        aco[2][2] = (aix * be - aiy * al) / d;
    }

    aco[0][0] = al;
    aco[1][0] = be;
    aco[2][0] = ge;

    //if (fabs(skk(aco[0][0], aco[1][0], aco[2][0], aco[0][1], aco[1][1], aco[2][1])) > 0.000001 || //
    //    fabs(skk(aco[0][0], aco[1][0], aco[2][0], aco[0][2], aco[1][2], aco[2][2])) > 0.000001 || //
    //    fabs(skk(aco[0][2], aco[1][2], aco[2][2], aco[0][1], aco[1][1], aco[2][1])) > 0.000001 || //
    //    fabs(kvv(aco[0][0], aco[1][0], aco[2][0]) - 1.0) > 0.000001 || fabs(kvv(aco[0][1], aco[1][1], aco[2][1]) - 1.0) > 0.000001 ||//
    //    fabs(kvv(aco[0][2], aco[1][2], aco[2][2]) - 1.0) > 0.000001)
    //{
    //    printf("Ne normal  174fdcdsaxes\n");
    //}


    for (int i = 0; i < 3; i++)
    {
        vL[i] = aco[0][i] * u1 + aco[1][i] * v1 + aco[2][i] * w1;
        vR[i] = aco[0][i] * u2 + aco[1][i] * v2 + aco[2][i] * w2;
        bL[i] = aco[0][i] * bx1 + aco[1][i] * by1 + aco[2][i] * bz1;
        bR[i] = aco[0][i] * bx2 + aco[1][i] * by2 + aco[2][i] * bz2;
    }



    double aaL = bL[0] / __dsqrt_rn(r1);
    double b2L = kv(bL[0]) + kv(bL[1]) + kv(bL[2]);
    double b21 = b2L / r1;
    double cL = __dsqrt_rn(ga * p1 / r1);
    double qp = __dsqrt_rn(b21 + cL * (cL + 2.0 * aaL));
    double qm = __dsqrt_rn(b21 + cL * (cL - 2.0 * aaL));
    double cfL = (qp + qm) / x2;
    double ptL = p1 + b2L / x2;

    double aaR = bR[0] / __dsqrt_rn(r2);
    double b2R = kv(bR[0]) + kv(bR[1]) + kv(bR[2]);
    double b22 = b2R / r2;
    double cR = __dsqrt_rn(ga * p2 / r2);
    qp = __dsqrt_rn(b22 + cR * (cR + 2.0 * aaR));
    qm = __dsqrt_rn(b22 + cR * (cR - 2.0 * aaR));
    double cfR = (qp + qm) / x2;
    double ptR = p2 + b2R / x2;

    double aC = (aaL + aaR) / x2;
    double b2o = (b22 + b21) / x2;
    double cC = __dsqrt_rn(ga * ap / ro);
    qp = __dsqrt_rn(b2o + cC * (cC + x2 * aC));
    qm = __dsqrt_rn(b2o + cC * (cC - x2 * aC));
    double cfC = (qp + qm) / x2;
    double vC1 = (vL[0] + vR[0]) / x2;

    double SL, SR;

    if (true)
    {
        SL = min(vL[0], vR[0]) - max(cfL, cfR);
        SR = max(vL[0], vR[0]) + max(cfL, cfR);
    }
    else if (n_disco == 1)
    {
        SL = min((vL[0] - cfL), (vC1 - cfC));
        SR = max((vR[0] + cfR), (vC1 + cfC));
    }
    else if (n_disco == 0)
    {
        SL = min((vL[0] - cfL), (vR[0] - cfR));
        SR = max((vL[0] + cfL), (vR[0] + cfR));
    }
    else if (n_disco == 2)
    {
        double SL_1 = min((vL[0] - cfL), (vC1 - cfC));
        double SR_1 = max((vR[0] + cfR), (vC1 + cfC));
        double SL_2 = min((vL[0] - cfL), (vR[0] - cfR));
        double SR_2 = max((vL[0] + cfL), (vR[0] + cfR));
        double oo = 0.75;
        double oo1 = 1.0 - oo;
        SL = oo * SL_1 + oo1 * SL_2;
        SR = oo * SR_1 + oo1 * SR_2;
    }


    double suR = SR - vR[0];
    double suL = SL - vL[0];
    double SM = (suR * r2 * vR[0] - ptR + ptL - suL * r1 * vL[0]) / (suR * r2 - suL * r1);

    // double dsl = SL;
    // double dsc = SM;
    // double dsp = SR;

    if ((SR < SL) || (SL > SM) || (SR < SM))
    {
        printf("ERROR -  254 fghrvtrgr\n");
        printf("%lf, %lf, %lf, %lf, %lf, %lf, %lf, %lf, %lf, %lf\n",//
            vL[0], vR[0], cfL, cfR, ro_L, ro_R, p_L, p_R, suR, suL);
    }

    double UU = max(fabs(SL), fabs(SR));
    double time = krit * rad / UU;

    double TR0, TL0;
    if (n_state == 0)
    {
        TR0 = fabs(vL[0] + vR[0]) / x2 + cfC;
        TL0 = -TR0;
        SR = TR0;
        SL = TL0;
    }


    double upt1 = (kv(u1) + kv(v1) + kv(w1)) / 2.0;
    double sbv1 = u1 * bx1 + v1 * by1 + w1 * bz1;

    double upt2 = (kv(u2) + kv(v2) + kv(w2)) / 2.0;
    double sbv2 = u2 * bx2 + v2 * by2 + w2 * bz2;

    double e1 = p1 / g1 + r1 * upt1 + b2L / x2;
    double e2 = p2 / g1 + r2 * upt2 + b2R / x2;
    double FQ_L, FQ_R;


    FQ_L = Q_L * vL[0];
    FL[0] = r1 * vL[0];
    FL[1] = r1 * vL[0] * vL[0] + ptL - kv(bL[0]);
    FL[2] = r1 * vL[0] * vL[1] - bL[0] * bL[1];
    FL[3] = r1 * vL[0] * vL[2] - bL[0] * bL[2];
    FL[4] = (e1 + ptL) * vL[0] - bL[0] * sbv1;
    FL[5] = 0.0;
    FL[6] = vL[0] * bL[1] - vL[1] * bL[0];
    FL[7] = vL[0] * bL[2] - vL[2] * bL[0];

    FQ_R = Q_R * vR[0];
    FR[0] = r2 * vR[0];
    FR[1] = r2 * vR[0] * vR[0] + ptR - kv(bR[0]);
    FR[2] = r2 * vR[0] * vR[1] - bR[0] * bR[1];
    FR[3] = r2 * vR[0] * vR[2] - bR[0] * bR[2];
    FR[4] = (e2 + ptR) * vR[0] - bR[0] * sbv2;
    FR[5] = 0.0;
    FR[6] = vR[0] * bR[1] - vR[1] * bR[0];
    FR[7] = vR[0] * bR[2] - vR[2] * bR[0];

    UL[0] = r1;
    UL[4] = e1;
    UR[0] = r2;
    UR[4] = e2;

    for (int i = 0; i < 3; i++)
    {

        UL[i + 1] = r1 * vL[i];
        UL[i + 5] = bL[i];
        UR[i + 1] = r2 * vR[i];
        UR[i + 5] = bR[i];
    }

    for (int ik = 0; ik < 8; ik++)
    {
        UZ[ik] = (SR * UR[ik] - SL * UL[ik] + FL[ik] - FR[ik]) / (SR - SL);
    }
    double UZQ = (SR * Q_R - SL * Q_L + FQ_L - FQ_R) / (SR - SL);

    if (n_state <= 1)
    {

        for (int ik = 0; ik < 8; ik++)
        {
            dq[ik] = UR[ik] - UL[ik];
        }
        double dqQ = Q_R - Q_L;



        double TL = SL;
        double TR = SR;
        if (SL > wv)
        {
            TL = 0.0;
            for (int ik = 0; ik < 8; ik++)
            {
                FW[ik] = wv * UL[ik];
            }
        }
        else if ((SL <= wv) && (wv <= SR))
        {
            for (int ik = 0; ik < 8; ik++)
            {
                FW[ik] = wv * UZ[ik];
            }
        }
        else if (SR < wv)
        {
            TR = 0.0;
            for (int ik = 0; ik < 8; ik++)
            {
                FW[ik] = wv * UR[ik];
            }
        }
        else
        {
            printf("ERROR  329 87732, %lf, %lf, %lf, %lf, %lf, %lf, %lf\n", r1, r2, p1, p2, al, be, ge);
        }


        double a = TR * TL;
        double b = TR - TL;

        PQ = (TR * FQ_L - TL * FQ_R + a * dqQ) / b;
        P[0] = (TR * FL[0] - TL * FR[0] + a * dq[0]) / b - FW[0];
        P[4] = (TR * FL[4] - TL * FR[4] + a * dq[4]) / b - FW[4];

        for (int ik = 1; ik < 4; ik++)
        {
            qv[ik - 1] = (TR * FL[ik] - TL * FR[ik] + a * dq[ik]) / b - FW[ik];
        }

        for (int ik = 5; ik < 8; ik++)
        {
            qb[ik - 5] = (TR * FL[ik] - TL * FR[ik] + a * dq[ik]) / b - FW[ik];
        }

        double SN = max(fabs(SL), fabs(SR));

        double wbn = 0.0;
        if (wv >= SR)
        {
            wbn = wv * bR[0];
        }
        else if (wv <= SL)
        {
            wbn = wv * bL[0];
        }
        else
        {
            wbn = wv * (bL[0] + bR[0]) / x2;
        }

        qb[0] = -SN * (bR[0] - bL[0]) - wbn;

        for (int ik = 0; ik < 3; ik++)
        {
            P[ik + 1] = aco[ik][0] * qv[0] + aco[ik][1] * qv[1] + aco[ik][2] * qv[2];
            P[ik + 5] = aco[ik][0] * qb[0] + aco[ik][1] * qb[1] + aco[ik][2] * qb[2];
            P[ik + 5] = spi4 * P[ik + 5];
        }

        double SWAP = P[4];
        P[4] = P[5];
        P[5] = P[6];
        P[6] = P[7];
        P[7] = SWAP;
        return time;
    }
    if (n_state == 3)
    {

        double ptz = (suR * r2 * ptL - suL * r1 * ptR + r1 * r2 * suR * suL * (vR[0] - vL[0])) / (suR * r2 - suL * r1);

        vzL[0] = SM;
        vzR[0] = SM;
        vzzL[0] = SM;
        vzzR[0] = SM;
        double ptzL = ptz;
        double ptzR = ptz;
        double ptzzL = ptz;
        double ptzzR = ptz;

        double suRm = suR / (SR - SM);
        double suLm = suL / (SL - SM);
        double rzR = r2 * suRm;
        double rzL = r1 * suLm;

        double bn = UZ[5];
        double bn2 = bn * bn;
        bzL[0] = bn;
        bzR[0] = bn;
        bzzL[0] = bn;
        bzzR[0] = bn;

        double ttR = r2 * suR * (SR - SM) - bn2;
        double tvR, tbR, tvL, tbL;
        if (fabs(ttR) <= 0.000001)
        {
            tvR = x0;
            tbR = x0;
        }
        else
        {
            tvR = (SM - vR[0]) / ttR;
            tbR = (r2 * suR * suR - bn2) / ttR;
        }

        double ttL = r1 * suL * (SL - SM) - bn2;
        if (fabs(ttL) <= 0.000001)
        {
            tvL = x0;
            tbL = x0;
        }
        else
        {
            tvL = (SM - vL[0]) / ttL;
            tbL = (r1 * suL * suL - bn2) / ttL;
        }

        vzL[1] = vL[1] - bn * bL[1] * tvL;
        vzL[2] = vL[2] - bn * bL[2] * tvL;
        vzR[1] = vR[1] - bn * bR[1] * tvR;
        vzR[2] = vR[2] - bn * bR[2] * tvR;

        bzL[1] = bL[1] * tbL;
        bzL[2] = bL[2] * tbL;
        bzR[1] = bR[1] * tbR;
        bzR[2] = bR[2] * tbR;

        double sbvL = bzL[0] * vzL[0] + bzL[1] * vzL[1] + bzL[2] * vzL[2];
        double sbvR = bzR[0] * vzR[0] + bzR[1] * vzR[1] + bzR[2] * vzR[2];

        double ezR = e2 * suRm + (ptz * SM - ptR * vR[0] + bn * (sbv2 - sbvR)) / (SR - SM);
        double ezL = e1 * suLm + (ptz * SM - ptL * vL[0] + bn * (sbv1 - sbvL)) / (SL - SM);

        double rzzR = rzR;
        double rzzL = rzL;
        double rzRs = __dsqrt_rn(rzR);
        double rzLs = __dsqrt_rn(rzL);
        double rzss = rzRs + rzLs;
        double rzps = rzRs * rzLs;

        double SZL = SM - fabs(bn) / rzLs;
        double SZR = SM + fabs(bn) / rzRs;

        int ibn = 0;
        double sbn;
        if (fabs(bn) > 0.000001)
        {
            sbn = 1.0 * sign(bn);
            ibn = 1;
        }
        else
        {
            sbn = 0.0;
            ibn = 0;
            SZL = SM;
            SZR = SM;
        }

        vzzL[1] = (rzLs * vzL[1] + rzRs * vzR[1] + sbn * (bzR[1] - bzL[1])) / rzss;
        vzzL[2] = (rzLs * vzL[2] + rzRs * vzR[2] + sbn * (bzR[2] - bzL[2])) / rzss;
        vzzR[1] = vzzL[1];
        vzzR[2] = vzzL[2];

        bzzL[1] = (rzLs * bzR[1] + rzRs * bzL[1] + sbn * rzps * (vzR[1] - vzL[1])) / rzss;
        bzzL[2] = (rzLs * bzR[2] + rzRs * bzL[2] + sbn * rzps * (vzR[2] - vzL[2])) / rzss;
        bzzR[1] = bzzL[1];
        bzzR[2] = bzzL[2];

        double sbzz = bzzL[0] * vzzL[0] + bzzL[1] * vzzL[1] + bzzL[2] * vzzL[2];

        double ezzR = ezR + rzRs * sbn * (sbvR - sbzz);
        double ezzL = ezL - rzLs * sbn * (sbvL - sbzz);

        UZL[0] = rzL;
        UZL[4] = ezL;
        UZR[0] = rzR;
        UZR[4] = ezR;

        for (int ik = 0; ik < 3; ik++)
        {
            UZL[ik + 1] = vzL[ik] * rzL;
            UZL[ik + 5] = bzL[ik];
            UZR[ik + 1] = vzR[ik] * rzR;
            UZR[ik + 5] = bzR[ik];
        }



        UZZL[0] = rzzL;
        UZZL[4] = ezzL;
        UZZR[0] = rzzR;
        UZZR[4] = ezzR;
        for (int ik = 0; ik < 3; ik++)
        {
            UZZL[ik + 1] = vzzL[ik] * rzzL;
            UZZL[ik + 5] = bzzL[ik];
            UZZR[ik + 1] = vzzR[ik] * rzzR;
            UZZR[ik + 5] = bzzR[ik];
        }



        if (SL > wv)
        {
            P[0] = FL[0] - wv * UL[0];
            PQ = P[0] * Q_L / r1;
            P[4] = FL[4] - wv * UL[4];
            for (int ik = 1; ik < 4; ik++)
            {
                qv[ik - 1] = FL[ik] - wv * UL[ik];
            }

            for (int ik = 5; ik < 8; ik++)
            {
                qb[ik - 5] = FL[ik] - wv * UL[ik];
            }
        }

        if ((SL <= wv) && (SZL >= wv))
        {
            int ik = 0;
            P[ik] = FL[ik] + SL * (UZL[ik] - UL[ik]) - wv * UZL[ik];
            PQ = P[ik] * Q_L / r1;
            ik = 4;
            P[ik] = FL[ik] + SL * (UZL[ik] - UL[ik]) - wv * UZL[ik];
            for (int ik = 1; ik < 4; ik++)
            {
                qv[ik - 1] = FL[ik] + SL * (UZL[ik] - UL[ik]) - wv * UZL[ik];
            }

            for (int ik = 5; ik < 8; ik++)
            {
                qb[ik - 5] = FL[ik] + SL * (UZL[ik] - UL[ik]) - wv * UZL[ik];
            }
        }
        //c------ FZZ
        if (ibn == 1)
        {

            if ((SZL <= wv) && (SM >= wv))
            {
                int ik = 0;
                P[ik] = FL[ik] + SZL * (UZZL[ik] - UZL[ik]) + SL * (UZL[ik] - UL[ik]) - wv * UZZL[ik];
                PQ = P[ik] * Q_L / r1;
                ik = 4;
                P[ik] = FL[ik] + SZL * (UZZL[ik] - UZL[ik]) + SL * (UZL[ik] - UL[ik]) - wv * UZZL[ik];
                for (int ik = 1; ik < 4; ik++)
                {
                    qv[ik - 1] = FL[ik] + SZL * (UZZL[ik] - UZL[ik]) + SL * (UZL[ik] - UL[ik]) - wv * UZZL[ik];
                }

                for (int ik = 5; ik < 8; ik++)
                {
                    qb[ik - 5] = FL[ik] + SZL * (UZZL[ik] - UZL[ik]) + SL * (UZL[ik] - UL[ik]) - wv * UZZL[ik];
                }
            }

            if ((SM <= wv) && (SZR >= wv))
            {
                int ik = 0;
                P[ik] = FR[ik] + SZR * (UZZR[ik] - UZR[ik]) + SR * (UZR[ik] - UR[ik]) - wv * UZZR[ik];
                PQ = P[ik] * Q_R / r2;
                ik = 4;
                P[ik] = FR[ik] + SZR * (UZZR[ik] - UZR[ik]) + SR * (UZR[ik] - UR[ik]) - wv * UZZR[ik];
                for (int ik = 1; ik < 4; ik++)
                {
                    qv[ik - 1] = FR[ik] + SZR * (UZZR[ik] - UZR[ik]) + SR * (UZR[ik] - UR[ik]) - wv * UZZR[ik];
                }

                for (int ik = 5; ik < 8; ik++)
                {
                    qb[ik - 5] = FR[ik] + SZR * (UZZR[ik] - UZR[ik]) + SR * (UZR[ik] - UR[ik]) - wv * UZZR[ik];
                }
            }

        }
        //c------ 
        if ((SZR <= wv) && (SR >= wv))
        {
            int ik = 0;
            P[ik] = FR[ik] + SR * (UZR[ik] - UR[ik]) - wv * UZR[ik];
            PQ = P[ik] * Q_R / r2;
            ik = 4;
            P[ik] = FR[ik] + SR * (UZR[ik] - UR[ik]) - wv * UZR[ik];

            for (int ik = 1; ik < 4; ik++)
            {
                qv[ik - 1] = FR[ik] + SR * (UZR[ik] - UR[ik]) - wv * UZR[ik];
            }

            for (int ik = 5; ik < 8; ik++)
            {
                qb[ik - 5] = FR[ik] + SR * (UZR[ik] - UR[ik]) - wv * UZR[ik];
            }
        }

        if (SR < wv)
        {
            P[0] = FR[0] - wv * UR[0];
            PQ = P[0] * Q_R / r2;
            P[4] = FR[4] - wv * UR[4];
            for (int ik = 1; ik < 4; ik++)
            {
                qv[ik - 1] = FR[ik] - wv * UR[ik];
            }


            for (int ik = 5; ik < 8; ik++)
            {
                qb[ik - 5] = FR[ik] - wv * UR[ik];
            }
        }


        //c----- Bn
        //double SN = max(fabs(SL), fabs(SR));

        double SN = max(fabs(SL), fabs(SR));

        double wbn = 0.0;
        if (wv >= SR)
        {
            wbn = wv * bR[0];
        }
        else if (wv <= SL)
        {
            wbn = wv * bL[0];
        }
        else
        {
            wbn = wv * (bL[0] + bR[0]) / x2;
        }

        qb[0] = -SN * (bR[0] - bL[0]) - wbn;

        //c-----


        for (int ik = 0; ik < 3; ik++)
        {
            P[ik + 1] = aco[ik][0] * qv[0] + aco[ik][1] * qv[1] + aco[ik][2] * qv[2];
            P[ik + 5] = aco[ik][0] * qb[0] + aco[ik][1] * qb[1] + aco[ik][2] * qb[2];
            P[ik + 5] = spi4 * P[ik + 5];
        }

        double SWAP = P[4];
        P[4] = P[5];
        P[5] = P[6];
        P[6] = P[7];
        P[7] = SWAP;
        return time;
    }
}


__device__ double HLLDQ_Korolkov(const double& ro_L, const double& Q_L, const double& p_L, const double& v1_L, const double& v2_L, const double& v3_L,//
    const double& Bx_L, const double& By_L, const double& Bz_L, const double& ro_R, const double& Q_R, const double& p_R, const double& v1_R, const double& v2_R, const double& v3_R,//
    const double& Bx_R, const double& By_R, const double& Bz_R, double* P, double& PQ, const double& n1, const double& n2, const double& n3, double& rad, int metod)
{// �� ��������, ���� �������� ����� �� �������
 // ������� ����� ��������� �� ���� ��������� ! (����� ����� ������� ������������)

        double bx_L = Bx_L / spi4;
        double by_L = By_L / spi4;
        double bz_L = Bz_L / spi4;

        double bx_R = Bx_R / spi4;
        double by_R = By_R / spi4;
        double bz_R = Bz_R / spi4;

        double t1 = 0.0;
        double t2 = 0.0;
        double t3 = 0.0;

        double m1 = 0.0;
        double m2 = 0.0;
        double m3 = 0.0;

        if (n1 > 0.1)
        {
            t2 = 1.0;
            m3 = 1.0;
        }
        else if (n2 > 0.1)
        {
            t3 = 1.0;
            m1 = 1.0;
        }
        else if (n3 > 0.1)
        {
            t1 = 1.0;
            m2 = 1.0;
        }
        else if (n1 < -0.1)
        {
            t3 = -1.0;
            m2 = -1.0;
        }
        else if (n2 < -0.1)
        {
            t1 = -1.0;
            m3 = -1.0;
        }
        else if (n3 < -0.1)
        {
            t1 = -1.0;
            m2 = -1.0;
        }
        else
        {
            printf("EROROR 1421  normal_error\n");
        }


        double u1, v1, w1, u2, v2, w2;
        u1 = v1_L * n1 + v2_L * n2 + v3_L * n3;
        v1 = v1_L * t1 + v2_L * t2 + v3_L * t3;
        w1 = v1_L * m1 + v2_L * m2 + v3_L * m3;
        u2 = v1_R * n1 + v2_R * n2 + v3_R * n3;
        v2 = v1_R * t1 + v2_R * t2 + v3_R * t3;
        w2 = v1_R * m1 + v2_R * m2 + v3_R * m3;

        double bn1, bt1, bm1, bn2, bt2, bm2;
        bn1 = bx_L * n1 + by_L * n2 + bz_L * n3;
        bt1 = bx_L * t1 + by_L * t2 + bz_L * t3;
        bm1 = bx_L * m1 + by_L * m2 + bz_L * m3;
        bn2 = bx_R * n1 + by_R * n2 + bz_R * n3;
        bt2 = bx_R * t1 + by_R * t2 + bz_R * t3;
        bm2 = bx_R * m1 + by_R * m2 + bz_R * m3;

        //cout << " = " << bt2 * bt2 + bm2 * bm2 << endl;

        double sqrtroL = sqrt(ro_L);
        double sqrtroR = sqrt(ro_R);
        double ca_L = bn1 / sqrtroL;
        double ca_R = bn2 / sqrtroR;
        double cL = sqrt(ggg * p_L / ro_L);
        double cR = sqrt(ggg * p_R / ro_R);

        double bb_L = kv(bx_L) + kv(by_L) + kv(bz_L);
        double bb_R = kv(bx_R) + kv(by_R) + kv(bz_R);

        double aL = (kv(bx_L) + kv(by_L) + kv(bz_L)) / ro_L;
        double aR = (kv(bx_L) + kv(by_L) + kv(bz_L)) / ro_L;

        double uu_L = (kv(v1_L) + kv(v2_L) + kv(v3_L)) / 2.0;
        double uu_R = (kv(v1_R) + kv(v2_R) + kv(v3_R)) / 2.0;

        double cfL = sqrt((ggg * p_L + bb_L + //
            sqrt(kv(ggg * p_L + bb_L) - 4.0 * ggg * p_L * kv(bn1))) / (2.0 * ro_L));
        double cfR = sqrt((ggg * p_R + bb_R + //
            sqrt(kv(ggg * p_R + bb_R) - 4.0 * ggg * p_R * kv(bn2))) / (2.0 * ro_R));


        double SL = min(u1, u2) - max(cfL, cfR);
        double SR = max(u1, u2) + max(cfL, cfR);

        double pTL = p_L + bb_L / 2.0;
        double pTR = p_R + bb_R / 2.0;

        double suR = (SR - u2);
        double suL = (SL - u1);

        double SM = (suR * ro_R * u2 - suL * ro_L * u1 - pTR + pTL) //
            / (suR * ro_R - suL * ro_L);

        double PTT = (suR * ro_R * pTL - suL * ro_L * pTR + ro_L * ro_R * suR * suL * (u2 - u1))//
            / (suR * ro_R - suL * ro_L);

        double UU = max(fabs(SL), fabs(SR));
        double time = krit * rad / UU;

        double FL[9], FR[9], UL[9], UR[9];

        double e1 = p_L / g1 + ro_L * uu_L + bb_L / 2.0;
        double e2 = p_R / g1 + ro_R * uu_R + bb_R / 2.0;


        FL[0] = ro_L * u1;
        FL[1] = ro_L * u1 * u1 + pTL - kv(bn1);
        FL[2] = ro_L * u1 * v1 - bn1 * bt1;
        FL[3] = ro_L * u1 * w1 - bn1 * bm1;
        FL[4] = (e1 + pTL) * u1 - bn1 * (u1 * bn1 + v1 * bt1 + w1 * bm1);
        //cout << uu_L << endl;
        FL[5] = 0.0;
        FL[6] = u1 * bt1 - v1 * bn1;
        FL[7] = u1 * bm1 - w1 * bn1;
        FL[8] = Q_L * u1;

        FR[0] = ro_R * u2;
        FR[1] = ro_R * u2 * u2 + pTR - kv(bn2);
        FR[2] = ro_R * u2 * v2 - bn2 * bt2;
        FR[3] = ro_R * u2 * w2 - bn2 * bm2;
        FR[4] = (e2 + pTR) * u2 - bn2 * (u2 * bn2 + v2 * bt2 + w2 * bm2);
        FR[5] = 0.0;
        FR[6] = u2 * bt2 - v2 * bn2;
        FR[7] = u2 * bm2 - w2 * bn2;
        FR[8] = Q_R * u2;

        UL[0] = ro_L;
        UL[1] = ro_L * u1;
        UL[2] = ro_L * v1;
        UL[3] = ro_L * w1;
        UL[4] = e1;
        UL[5] = bn1;
        UL[6] = bt1;
        UL[7] = bm1;
        UL[8] = Q_L;

        UR[0] = ro_R;
        UR[1] = ro_R * u2;
        UR[2] = ro_R * v2;
        UR[3] = ro_R * w2;
        UR[4] = e2;
        UR[5] = bn2;
        UR[6] = bt2;
        UR[7] = bm2;
        UR[8] = Q_R;

        double bn = (SR * UR[5] - SL * UL[5] + FL[5] - FR[5]) / (SR - SL);
        double bt = (SR * UR[6] - SL * UL[6] + FL[6] - FR[6]) / (SR - SL);
        double bm = (SR * UR[7] - SL * UL[7] + FL[7] - FR[7]) / (SR - SL);
        double bbn = bn * bn;

        double ro_LL = ro_L * (SL - u1) / (SL - SM);
        double ro_RR = ro_R * (SR - u2) / (SR - SM);
        double Q_LL = Q_L * (SL - u1) / (SL - SM);
        double Q_RR = Q_R * (SR - u2) / (SR - SM);

        if (metod == 2)   // HLLC  + mgd
        {
            double sbv1 = u1 * bn1 + v1 * bt1 + w1 * bm1;
            double sbv2 = u2 * bn2 + v2 * bt2 + w2 * bm2;

            double UZ0 = (SR * UR[0] - SL * UL[0] + FL[0] - FR[0]) / (SR - SL);
            double UZ1 = (SR * UR[1] - SL * UL[1] + FL[1] - FR[1]) / (SR - SL);
            double UZ2 = (SR * UR[2] - SL * UL[2] + FL[2] - FR[2]) / (SR - SL);
            double UZ3 = (SR * UR[3] - SL * UL[3] + FL[3] - FR[3]) / (SR - SL);
            double UZ4 = (SR * UR[4] - SL * UL[4] + FL[4] - FR[4]) / (SR - SL);
            double vzL, vzR, vLL, wLL, vRR, wRR, ppLR, btt1, bmm1, btt2, bmm2, ee1, ee2;


            double suRm = suR / (SR - SM);
            double suLm = suL / (SL - SM);
            double rzR = ro_R * suRm;
            double rzL = ro_L * suLm;

            double ptzR = pTR + ro_R * suR * (SM - u2);
            double ptzL = pTL + ro_L * suL * (SM - u1);
            double ptz = (ptzR + ptzL) / 2.0;


            vRR = UZ2 / UZ0;
            wRR = UZ3 / UZ0;
            vLL = vRR;
            wLL = wRR;

            /*vRR = v2 + bn * (bt2 - bt) / suR / ro_R;
            wRR = w2 + bn * (bm2 - bm) / suR / ro_R;
            vLL = v1 + bn * (bt1 - bt) / suL / ro_L;
            wLL = w1 + bn * (bm1 - bm) / suL / ro_L;*/

            btt2 = bt;
            bmm2 = bm;
            btt1 = btt2;
            bmm1 = bmm2;

            double sbvz = (bn * UZ1 + bt * UZ2 + bm * UZ3) / UZ0;

            ee2 = e2 * suRm + (ptz * SM - pTR * u2 + bn * (sbv2 - sbvz)) / (SR - SM);
            ee1 = e1 * suLm + (ptz * SM - pTL * u1 + bn * (sbv1 - sbvz)) / (SL - SM);

            /*if (fabs(bn) < 0.000001 )
            {
                vRR = v2;
                wRR = w2;
                vLL = v1;
                wLL = w1;
                btt2 = bt2 * suRm;
                bmm2 = bm2 * suRm;
                btt1 = bt1 * suLm;
                bmm1 = bm1 * suLm;
            }*/

            /*ppLR = (pTL + ro_L * (SL - u1) * (SM - u1) + pTR + ro_R * (SR - u2) * (SM - u2)) / 2.0;

            if (fabs(bn) < 0.000001)
            {
                vLL = v1;
                wLL = w1;
                vRR = v2;
                wRR = w2;

                btt1 = bt1 * (SL - u1) / (SL - SM);
                btt2 = bt2 * (SR - u2) / (SR - SM);

                bmm1 = bm1 * (SL - u1) / (SL - SM);
                bmm2 = bm2 * (SR - u2) / (SR - SM);

                ee1 = ((SL - u1) * e1 - pTL * u1 + ppLR * SM) / (SL - SM);
                ee2 = ((SR - u2) * e2 - pTL * u2 + ppLR * SM) / (SR - SM);
            }
            else
            {
                btt2 = btt1 = (SR * UR[6] - SL * UL[6] + FL[6] - FR[6]) / (SR - SL);
                bmm2 = bmm1 = (SR * UR[7] - SL * UL[7] + FL[7] - FR[7]) / (SR - SL);
                vLL = v1 + bn * (bt1 - btt1) / (ro_L * (SL - u1));
                vRR = v2 + bn * (bt2 - btt2) / (ro_R * (SR - u2));

                wLL = w1 + bn * (bm1 - bmm1) / (ro_L * (SL - u1));
                wRR = w2 + bn * (bm2 - bmm2) / (ro_R * (SR - u2));

                double sks1 = u1 * bn1 + v1 * bt1 + w1 * bm1 - SM * bn - vLL * btt1 - wLL * bmm1;
                double sks2 = u2 * bn2 + v2 * bt2 + w2 * bm2 - SM * bn - vRR * btt2 - wRR * bmm2;

                ee1 = ((SL - u1) * e1 - pTL * u1 + ppLR * SM + bn * sks1) / (SL - SM);
                ee2 = ((SR - u2) * e2 - pTR * u2 + ppLR * SM + bn * sks2) / (SR - SM);
            }*/


            double  ULL[9], URR[9], PO[9];
            ULL[0] = ro_LL;
            ULL[1] = ro_LL * SM;
            ULL[2] = ro_LL * vLL;
            ULL[3] = ro_LL * wLL;
            ULL[4] = ee1;
            ULL[5] = bn;
            ULL[6] = btt1;
            ULL[7] = bmm1;
            ULL[8] = Q_LL;

            URR[0] = ro_RR;
            URR[1] = ro_RR * SM;
            URR[2] = ro_RR * vRR;
            URR[3] = ro_RR * wRR;
            URR[4] = ee2;
            URR[5] = bn;
            URR[6] = btt2;
            URR[7] = bmm2;
            URR[8] = Q_RR;

            if (SL >= 0.0)
            {
                for (int i = 0; i < 9; i++)
                {
                    PO[i] = FL[i];
                }
            }
            else if (SL < 0.0 && SM >= 0.0)
            {
                for (int i = 0; i < 9; i++)
                {
                    PO[i] = FL[i] + SL * ULL[i] - SL * UL[i];
                }
            }
            else if (SR > 0.0 && SM < 0.0)
            {
                for (int i = 0; i < 9; i++)
                {
                    PO[i] = FR[i] + SR * URR[i] - SR * UR[i];
                }
            }
            else if (SR <= 0.0)
            {
                for (int i = 0; i < 9; i++)
                {
                    PO[i] = FR[i];
                }
            }



            double SN = max(fabs(SL), fabs(SR));

            PO[5] = -SN * (bn2 - bn1);

            P[1] = n1 * PO[1] + t1 * PO[2] + m1 * PO[3];
            P[2] = n2 * PO[1] + t2 * PO[2] + m2 * PO[3];
            P[3] = n3 * PO[1] + t3 * PO[2] + m3 * PO[3];
            P[5] = spi4 * (n1 * PO[5] + t1 * PO[6] + m1 * PO[7]);
            P[6] = spi4 * (n2 * PO[5] + t2 * PO[6] + m2 * PO[7]);
            P[7] = spi4 * (n3 * PO[5] + t3 * PO[6] + m3 * PO[7]);
            P[0] = PO[0];
            P[4] = PO[4];
            PQ = PO[8];

            double SWAP = P[4];
            P[4] = P[5];
            P[5] = P[6];
            P[6] = P[7];
            P[7] = SWAP;
            return time;

        }
        else if (metod == 3)  // HLLD
        {

            double ttL = ro_L * suL * (SL - SM) - bbn;
            double ttR = ro_R * suR * (SR - SM) - bbn;

            double vLL, wLL, vRR, wRR, btt1, bmm1, btt2, bmm2;

            if (fabs(ttL) >= 0.000001)
            {
                vLL = v1 - bn * bt1 * (SM - u1) / ttL;
                wLL = w1 - bn * bm1 * (SM - u1) / ttL;
                btt1 = bt1 * (ro_L * suL * suL - bbn) / ttL;
                bmm1 = bm1 * (ro_L * suL * suL - bbn) / ttL;
            }
            else
            {
                vLL = v1;
                wLL = w1;
                btt1 = 0.0;
                bmm1 = 0.0;
            }

            if (fabs(ttR) >= 0.000001)
            {
                vRR = v2 - bn * bt2 * (SM - u2) / ttR;
                wRR = w2 - bn * bm2 * (SM - u2) / ttR;
                btt2 = bt2 * (ro_R * suR * suR - bbn) / ttR;
                bmm2 = bm2 * (ro_R * suR * suR - bbn) / ttR;
                //cout << "tbr = " << (ro_R * suR * suR - bbn) / ttR << endl;
                //cout << "bt2 = " << bt2 << endl;
            }
            else
            {
                vRR = v2;
                wRR = w2;
                btt2 = 0.0;
                bmm2 = 0.0;
            }

            double eLL = (e1 * suL + PTT * SM - pTL * u1 + bn * //
                ((u1 * bn1 + v1 * bt1 + w1 * bm1) - (SM * bn + vLL * btt1 + wLL * bmm1))) //
                / (SL - SM);
            double eRR = (e2 * suR + PTT * SM - pTR * u2 + bn * //
                ((u2 * bn2 + v2 * bt2 + w2 * bm2) - (SM * bn + vRR * btt2 + wRR * bmm2))) //
                / (SR - SM);

            double sqrtroLL = sqrt(ro_LL);
            double sqrtroRR = sqrt(ro_RR);
            double SLL = SM - fabs(bn) / sqrtroLL;
            double SRR = SM + fabs(bn) / sqrtroRR;

            double idbn = 1.0;
            if (fabs(bn) > 0.0001)
            {
                idbn = 1.0 * sign(bn);
            }
            else
            {
                idbn = 0.0;
                SLL = SM;
                SRR = SM;
            }

            double vLLL = (sqrtroLL * vLL + sqrtroRR * vRR + //
                idbn * (btt2 - btt1)) / (sqrtroLL + sqrtroRR);

            double wLLL = (sqrtroLL * wLL + sqrtroRR * wRR + //
                idbn * (bmm2 - bmm1)) / (sqrtroLL + sqrtroRR);

            double bttt = (sqrtroLL * btt2 + sqrtroRR * btt1 + //
                idbn * sqrtroLL * sqrtroRR * (vRR - vLL)) / (sqrtroLL + sqrtroRR);

            double bmmm = (sqrtroLL * bmm2 + sqrtroRR * bmm1 + //
                idbn * sqrtroLL * sqrtroRR * (wRR - wLL)) / (sqrtroLL + sqrtroRR);

            double eLLL = eLL - idbn * sqrtroLL * ((SM * bn + vLL * btt1 + wLL * bmm1) //
                - (SM * bn + vLLL * bttt + wLLL * bmmm));
            double eRRR = eRR + idbn * sqrtroRR * ((SM * bn + vRR * btt2 + wRR * bmm2) //
                - (SM * bn + vLLL * bttt + wLLL * bmmm));
            //cout << " = " << bn << " " << btt2 << " " << bmm2 << endl;
            //cout << "sbvr = " << (SM * bn + vRR * btt2 + wRR * bmm2) << endl;
            double  ULL[9], URR[9], ULLL[9], URRR[9];

            ULL[0] = ro_LL;
            ULL[1] = ro_LL * SM;
            ULL[2] = ro_LL * vLL;
            ULL[3] = ro_LL * wLL;
            ULL[4] = eLL;
            ULL[5] = bn;
            ULL[6] = btt1;
            ULL[7] = bmm1;
            ULL[8] = Q_LL;

            URR[0] = ro_RR;
            //cout << ro_RR << endl;
            URR[1] = ro_RR * SM;
            URR[2] = ro_RR * vRR;
            URR[3] = ro_RR * wRR;
            URR[4] = eRR;
            URR[5] = bn;
            URR[6] = btt2;
            URR[7] = bmm2;
            URR[8] = Q_RR;

            ULLL[0] = ro_LL;
            ULLL[1] = ro_LL * SM;
            ULLL[2] = ro_LL * vLLL;
            ULLL[3] = ro_LL * wLLL;
            ULLL[4] = eLLL;
            ULLL[5] = bn;
            ULLL[6] = bttt;
            ULLL[7] = bmmm;
            ULLL[8] = Q_LL;

            URRR[0] = ro_RR;
            URRR[1] = ro_RR * SM;
            URRR[2] = ro_RR * vLLL;
            URRR[3] = ro_RR * wLLL;
            URRR[4] = eRRR;
            URRR[5] = bn;
            URRR[6] = bttt;
            URRR[7] = bmmm;
            URRR[8] = Q_RR;

            double PO[9];

            if (SL >= 0.0)
            {
                //cout << "SL >= 0.0" << endl;
                for (int i = 0; i < 9; i++)
                {
                    PO[i] = FL[i];
                }
            }
            else if (SL < 0.0 && SLL >= 0.0)
            {
                //cout << "SL < 0.0 && SLL >= 0.0" << endl;
                for (int i = 0; i < 9; i++)
                {
                    PO[i] = FL[i] + SL * ULL[i] - SL * UL[i];
                }
                //cout << ULL[0] << endl;
            }
            else if (SLL <= 0.0 && SM >= 0.0)
            {
                //cout << "SLL <= 0.0 && SM >= 0.0" << endl;
                for (int i = 0; i < 9; i++)
                {
                    PO[i] = FL[i] + SLL * ULLL[i] - (SLL - SL) * ULL[i] - SL * UL[i];
                }
            }
            else if (SM < 0.0 && SRR > 0.0)
            {
                //cout << "SM < 0.0 && SRR > 0.0" << endl;
                for (int i = 0; i < 9; i++)
                {
                    PO[i] = FR[i] + SRR * URRR[i] - (SRR - SR) * URR[i] - SR * UR[i];
                }
                //cout << "P4 = " << URRR[4] << endl;
            }
            else if (SR > 0.0 && SRR <= 0.0)
            {
                //cout << "SR > 0.0 && SRR <= 0.0" << endl;
                for (int i = 0; i < 9; i++)
                {
                    PO[i] = FR[i] + SR * URR[i] - SR * UR[i];
                }
                //cout << URR[0] << endl;
            }
            else if (SR <= 0.0)
            {
                //cout << "SR <= 0.0" << endl;
                for (int i = 0; i < 9; i++)
                {
                    PO[i] = FR[i];
                }
            }



            double SN = max(fabs(SL), fabs(SR));

            PO[5] = -SN * (bn2 - bn1);

            P[1] = n1 * PO[1] + t1 * PO[2] + m1 * PO[3];
            P[2] = n2 * PO[1] + t2 * PO[2] + m2 * PO[3];
            P[3] = n3 * PO[1] + t3 * PO[2] + m3 * PO[3];
            P[5] = spi4 * (n1 * PO[5] + t1 * PO[6] + m1 * PO[7]);
            P[6] = spi4 * (n2 * PO[5] + t2 * PO[6] + m2 * PO[7]);
            P[7] = spi4 * (n3 * PO[5] + t3 * PO[6] + m3 * PO[7]);
            P[0] = PO[0];
            P[4] = PO[4];
            PQ = PO[8];

            double SWAP = P[4];
            P[4] = P[5];
            P[5] = P[6];
            P[6] = P[7];
            P[7] = SWAP;
            return time;
        }

    }


/*{   // �� ��������, ���� �������� ����� �� �������
    // ������� ����� ��������� �� ���� ���������

    double bx_L = Bx_L / spi4;
    double by_L = By_L / spi4;
    double bz_L = Bz_L / spi4;

    double bx_R = Bx_R / spi4;
    double by_R = By_R / spi4;
    double bz_R = Bz_R / spi4;

    double t1 = 0.0;
    double t2 = 0.0;
    double t3 = 0.0;

    double m1 = 0.0;
    double m2 = 0.0;
    double m3 = 0.0;

    if (n1 > 0.1)
    {
        t2 = 1.0;
        m3 = 1.0;
    }
    else if (n2 > 0.1)
    {
        t3 = 1.0;
        m1 = 1.0;
    }
    else if (n3 > 0.1)
    {
        t1 = 1.0;
        m2 = 1.0;
    }
    else if (n1 < -0.1)
    {
        t3 = -1.0;
        m2 = -1.0;
    }
    else if (n2 < -0.1)
    {
        t1 = -1.0;
        m3 = -1.0;
    }
    else if (n3 < -0.1)
    {
        t1 = -1.0;
        m2 = -1.0;
    }
    else
    {
        printf("EROROR 1421  normal_error\n");
    }


    double u1, v1, w1, u2, v2, w2;
    u1 = v1_L * n1 + v2_L * n2 + v3_L * n3;
    v1 = v1_L * t1 + v2_L * t2 + v3_L * t3;
    w1 = v1_L * m1 + v2_L * m2 + v3_L * m3;
    u2 = v1_R * n1 + v2_R * n2 + v3_R * n3;
    v2 = v1_R * t1 + v2_R * t2 + v3_R * t3;
    w2 = v1_R * m1 + v2_R * m2 + v3_R * m3;

    double bn1, bt1, bm1, bn2, bt2, bm2;
    bn1 = bx_L * n1 + by_L * n2 + bz_L * n3;
    bt1 = bx_L * t1 + by_L * t2 + bz_L * t3;
    bm1 = bx_L * m1 + by_L * m2 + bz_L * m3;
    bn2 = bx_R * n1 + by_R * n2 + bz_R * n3;
    bt2 = bx_R * t1 + by_R * t2 + bz_R * t3;
    bm2 = bx_R * m1 + by_R * m2 + bz_R * m3;

    double sqrtroL = sqrt(ro_L);
    double sqrtroR = sqrt(ro_R);
    double ca_L = bn1 / sqrtroL;
    double ca_R = bn2 / sqrtroR;
    double cL = sqrt(ggg * p_L / ro_L);
    double cR = sqrt(ggg * p_R / ro_R);

    double bb_L = kv(bx_L) + kv(by_L) + kv(bz_L);
    double bb_R = kv(bx_R) + kv(by_R) + kv(bz_R);

    double aL = (kv(bx_L) + kv(by_L) + kv(bz_L)) / ro_L;
    double aR = (kv(bx_L) + kv(by_L) + kv(bz_L)) / ro_L;

    double uu_L = (kv(v1_L) + kv(v2_L) + kv(v3_L)) / 2.0;
    double uu_R = (kv(v1_R) + kv(v2_R) + kv(v3_R)) / 2.0;

    //double cfL = sqrt((ggg * p_L + bb_L + //
    //    sqrt(kv(ggg * p_L + bb_L) - 4.0 * ggg * p_L * kv(bn1))) / (2.0 * ro_L));
    //double cfR = sqrt((ggg * p_R + bb_R + //
    //    sqrt(kv(ggg * p_R + bb_R) - 4.0 * ggg * p_L * kv(bn2))) / (2.0 * ro_R));

    //double cfL = sqrt((kv(cL) + kv(aL)) / 2.0 + 0.5 * sqrt(kv(kv(cL) + kv(aL)) - 4.0 * kv(cL) * kv(ca_L)));
    //double cfR = sqrt((kv(cR) + kv(aR)) / 2.0 + 0.5 * sqrt(kv(kv(cR) + kv(aR)) - 4.0 * kv(cR) * kv(ca_R)));

    double aaL = bn1 / sqrt(ro_L);
    double b2L = kv(bn1) + kv(bt1) + kv(bm1);
    double b21 = b2L / ro_L;
    //double cL = sqrt(ga * p_L / ro_L);
    double qp = sqrt(b21 + cL * (cL + 2.0 * aaL));
    double qm = sqrt(b21 + cL * (cL - 2.0 * aaL));
    double cfL = (qp + qm) / 2.0;

    double aaR = bn2 / sqrt(ro_R);
    double b2R = kv(bn2) + kv(bt2) + kv(bm2);
    double b22 = b2R / ro_R;
    //double cR = sqrt(ga * p_R / ro_R);
    qp = sqrt(b22 + cR * (cR + 2.0 * aaR));
    qm = sqrt(b22 + cR * (cR - 2.0 * aaR));
    double cfR = (qp + qm) / 2.0;


    //cout << "cfR = " << cfR << " " << bn2 << " " << bt2 << " " << bm2 << endl;
    double SL = min(u1, u2) - max(cfL, cfR);
    double SR = max(u1, u2) + max(cfL, cfR);

    double pTL = p_L + bb_L / 2.0;
    double pTR = p_R + bb_R / 2.0;

    double suR = (SR - u2);
    double suL = (SL - u1);

    double SM = (suR * ro_R * u2 - suL * ro_L * u1 - pTR + pTL) //
        / (suR * ro_R - suL * ro_L);

    double PTT = (suR * ro_R * pTL - suL * ro_L * pTR + ro_L * ro_R * suR * suL * (u2 - u1))//
        / (suR * ro_R - suL * ro_L);

    double UU = max(fabs(SL), fabs(SR));
    double time = krit * rad / UU;

    double FL[9], FR[9], UL[9], UR[9];

    double e1 = p_L / g1 + ro_L * uu_L + bb_L / 2.0;
    double e2 = p_R / g1 + ro_R * uu_R + bb_R / 2.0;


    FL[0] = ro_L * u1;
    FL[1] = ro_L * u1 * u1 + pTL - kv(bn1);
    FL[2] = ro_L * u1 * v1 - bn1 * bt1;
    FL[3] = ro_L * u1 * w1 - bn1 * bm1;
    FL[4] = (e1 + pTL) * u1 - bn1 * (u1 * bn1 + v1 * bt1 + w1 * bm1);
    FL[5] = 0.0;
    FL[6] = u1 * bt1 - v1 * bn1;
    FL[7] = u1 * bm1 - w1 * bn1;
    FL[8] = Q_L * u1;

    FR[0] = ro_R * u2;
    FR[1] = ro_R * u2 * u2 + pTR - kv(bn2);
    FR[2] = ro_R * u2 * v2 - bn2 * bt2;
    FR[3] = ro_R * u2 * w2 - bn2 * bm2;
    FR[4] = (e2 + pTR) * u2 - bn2 * (u2 * bn2 + v2 * bt2 + w2 * bm2);
    FR[5] = 0.0;
    FR[6] = u2 * bt2 - v2 * bn2;
    FR[7] = u2 * bm2 - w2 * bn2;
    FR[8] = Q_R * u2;

    UL[0] = ro_L;
    UL[1] = ro_L * u1;
    UL[2] = ro_L * v1;
    UL[3] = ro_L * w1;
    UL[4] = e1;
    UL[5] = bn1;
    UL[6] = bt1;
    UL[7] = bm1;
    UL[8] = Q_L;

    UR[0] = ro_R;
    UR[1] = ro_R * u2;
    UR[2] = ro_R * v2;
    UR[3] = ro_R * w2;
    UR[4] = e2;
    UR[5] = bn2;
    UR[6] = bt2;
    UR[7] = bm2;
    UR[8] = Q_R;

    double bn = (SR * UR[5] - SL * UL[5] + FL[5] - FR[5]) / (SR - SL);
    double bbn = bn * bn;

    double ro_LL = ro_L * (SL - u1) / (SL - SM);
    double ro_RR = ro_R * (SR - u2) / (SR - SM);
    double Q_LL = Q_L * (SL - u1) / (SL - SM);
    double Q_RR = Q_R * (SR - u2) / (SR - SM);


    double ttL = ro_L * suL * (SL - SM) - bbn;
    double ttR = ro_R * suR * (SR - SM) - bbn;

    double vLL, wLL, vRR, wRR, btt1, bmm1, btt2, bmm2;

    if (fabs(ttL) >= 0.0000001)
    {
        vLL = v1 - bn * bt1 * (SM - u1) / ttL;
        wLL = w1 - bn * bm1 * (SM - u1) / ttL;
        btt1 = bt1 * (ro_L * suL * suL - bbn) / ttL;
        bmm1 = bm1 * (ro_L * suL * suL - bbn) / ttL;
    }
    else
    {
        vLL = v1;
        wLL = w1;
        btt1 = 0.0;
        bmm1 = 0.0;
    }

    if (fabs(ttR) >= 0.0000001)
    {
        vRR = v2 - bn * bt2 * (SM - u2) / ttR;
        wRR = w2 - bn * bm2 * (SM - u2) / ttR;
        btt2 = bt2 * (ro_R * suR * suR - bbn) / ttR;
        bmm2 = bm2 * (ro_R * suR * suR - bbn) / ttR;
    }
    else
    {
        vRR = v2;
        wRR = w2;
        btt2 = 0.0;
        bmm2 = 0.0;
    }

    double eLL = (e1 * suL + PTT * SM - pTL * u1 + bn * //
        ((u1 * bn1 + v1 * bt1 + w1 * bm1) - (SM * bn + vLL * btt1 + wLL * bmm1))) //
        / (SL - SM);
    double eRR = (e2 * suR + PTT * SM - pTR * u2 + bn * //
        ((u2 * bn2 + v2 * bt2 + w2 * bm2) - (SM * bn + vRR * btt2 + wRR * bmm2))) //
        / (SR - SM);

    double sqrtroLL = sqrt(ro_LL);
    double sqrtroRR = sqrt(ro_RR);
    double SLL = SM - fabs(bn) / sqrtroLL;
    double SRR = SM + fabs(bn) / sqrtroRR;

    double idbn = 1.0;
    if (fabs(bn) > 0.000001)
    {
        idbn = 1.0 * sign(bn);
    }
    else
    {
        idbn = 0.0;
        SLL = SM;
        SRR = SM;
    }

    double vLLL = (sqrtroLL * vLL + sqrtroRR * vRR + //
        idbn * (btt2 - btt1)) / (sqrtroLL + sqrtroRR);

    double wLLL = (sqrtroLL * wLL + sqrtroRR * wRR + //
        idbn * (bmm2 - bmm1)) / (sqrtroLL + sqrtroRR);

    double bttt = (sqrtroLL * btt2 + sqrtroRR * btt1 + //
        idbn * sqrtroLL * sqrtroRR * (vRR - vLL)) / (sqrtroLL + sqrtroRR);

    double bmmm = (sqrtroLL * bmm2 + sqrtroRR * bmm1 + //
        idbn * sqrtroLL * sqrtroRR * (wRR - wLL)) / (sqrtroLL + sqrtroRR);

    double eLLL = eLL - idbn * sqrtroLL * ((SM * bn + vLL * btt1 + wLL * bmm1) //
        - (SM * bn + vLLL * bttt + wLLL * bmmm));
    double eRRR = eRR + idbn * sqrtroRR * ((SM * bn + vRR * btt2 + wRR * bmm2) //
        - (SM * bn + vLLL * bttt + wLLL * bmmm));

    double  ULL[9], URR[9], ULLL[9], URRR[9];

    ULL[0] = ro_LL;
    ULL[1] = ro_LL * SM;
    ULL[2] = ro_LL * vLL;
    ULL[3] = ro_LL * wLL;
    ULL[4] = eLL;
    ULL[5] = bn;
    ULL[6] = btt1;
    ULL[7] = bmm1;
    ULL[8] = Q_LL;

    URR[0] = ro_RR;
    URR[1] = ro_RR * SM;
    URR[2] = ro_RR * vRR;
    URR[3] = ro_RR * wRR;
    URR[4] = eRR;
    URR[5] = bn;
    URR[6] = btt2;
    URR[7] = bmm2;
    URR[8] = Q_RR;

    ULLL[0] = ro_LL;
    ULLL[1] = ro_LL * SM;
    ULLL[2] = ro_LL * vLLL;
    ULLL[3] = ro_LL * wLLL;
    ULLL[4] = eLLL;
    ULLL[5] = bn;
    ULLL[6] = bttt;
    ULLL[7] = bmmm;
    ULLL[8] = Q_LL;

    URRR[0] = ro_RR;
    URRR[1] = ro_RR * SM;
    URRR[2] = ro_RR * vLLL;
    URRR[3] = ro_RR * wLLL;
    URRR[4] = eRRR;
    URRR[5] = bn;
    URRR[6] = bttt;
    URRR[7] = bmmm;
    URRR[8] = Q_RR;

    double PO[9];

    if (SL >= 0.0)
    {
        for (int i = 0; i < 9; i++)
        {
            PO[i] = FL[i];
        }
    }
    else if (SL < 0.0 && SLL >= 0.0)
    {
        for (int i = 0; i < 9; i++)
        {
            PO[i] = FL[i] + SL * ULL[i] - SL * UL[i];
        }
    }
    else if (SLL <= 0.0 && SM >= 0.0)
    {
        for (int i = 0; i < 9; i++)
        {
            PO[i] = FL[i] + SLL * ULLL[i] - (SLL - SL) * ULL[i] - SL * UL[i];
        }
    }
    else if (SM < 0.0 && SRR > 0.0)
    {
        for (int i = 0; i < 9; i++)
        {
            PO[i] = FR[i] + SRR * URRR[i] - (SRR - SR) * URR[i] - SR * UR[i];
        }
    }
    else if (SR > 0.0 && SRR <= 0.0)
    {
        for (int i = 0; i < 9; i++)
        {
            PO[i] = FR[i] + SR * URR[i] - SR * UR[i];
        }
    }
    else if (SR <= 0.0)
    {
        for (int i = 0; i < 9; i++)
        {
            PO[i] = FR[i];
        }
    }



    double SN = max(fabs(SL), fabs(SR));

    PO[5] = -SN * (bn2 - bn1);

    P[1] = n1 * PO[1] + t1 * PO[2] + m1 * PO[3];
    P[2] = n2 * PO[1] + t2 * PO[2] + m2 * PO[3];
    P[3] = n3 * PO[1] + t3 * PO[2] + m3 * PO[3];
    P[5] = spi4 * (n1 * PO[5] + t1 * PO[6] + m1 * PO[7]);
    P[6] = spi4 * (n2 * PO[5] + t2 * PO[6] + m2 * PO[7]);
    P[7] = spi4 * (n3 * PO[5] + t3 * PO[6] + m3 * PO[7]);
    P[0] = PO[0];
    PQ = PO[8];

    double SWAP = P[4];
    P[4] = P[5];
    P[5] = P[6];
    P[6] = P[7];
    P[7] = SWAP;
    return time;

}*/